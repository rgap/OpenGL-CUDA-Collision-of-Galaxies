#include "hip/hip_runtime.h"
#include "GalaxiaQT.cuh"

#include <iostream>

#include <hip/hip_runtime.h>
#include <>


using namespace std;

//Nbody
double Pxi,Pyi,Pzi,Fx,Fy,Fz,Mi;
double s,d,dx,dy,dz,dist2,dist,Fs,Fsdx,Fsdy,Fsdz,Gdtm;
int contador=0;
double Gdt;
double dt;

GalaxiaQT::GalaxiaQT(){
	
	//this->MaxNumCuerpos=(81920);

	//int numElem=35000;

	int numElem=81920;

	this->MaxNumCuerpos=(numElem);
	this->n_Cuerpos=0;

	//l_Cuerpos=new Cuerpo *[numElem];
	//l_Cuerpos=(Cuerpo**)malloc(sizeof(Cuerpo*)*numElem);

	Root=NULL;
	
}

void GalaxiaQT::addCuerpo(Cuerpo *cuerp){
	if (this->n_Cuerpos<MaxNumCuerpos){
		l_Cuerpos[this->n_Cuerpos] = *cuerp;
		n_Cuerpos++;				
	}
}


void GalaxiaQT::ObtenerHijo(Cuerpo *P,Cuerpo* H) {

	/*		0	|	1	|	2	|	3	|	4	|	5	|	6	|	7
			NE1	   SE1     SO1     NO1	   NE2	   SE2	    SO2	   NO2
	*/

	if(P->PosX<H->PosX){
		
		if(P->PosY<H->PosY){
			if(P->PosZ<H->PosZ){
				P->NE1=addCuerpo(P->NE1,H);	//0
			}else{
				P->NE2=addCuerpo(P->NE2,H);	//4
			}
		}else{
			if (P->PosZ<H->PosZ){
				P->SE1=addCuerpo(P->SE1,H);	//1	
			}else{
				P->SE2=addCuerpo(P->SE2,H);	//5	
			}
		}
	}else{
		if(P->PosY<H->PosY){
			if(P->PosZ<H->PosZ){
				P->NO1=addCuerpo(P->NO1,H);	//3		
			}else{
				P->NO2=addCuerpo(P->NO2,H);	//7
			}  
		}else{
			if (P->PosZ<H->PosZ){
				P->SO1=addCuerpo(P->SO1,H);	//2
			}else{
				P->SO2=addCuerpo(P->SO2,H);	//6
			}
		}
	}
}



Cuerpo* GalaxiaQT::addCuerpo(Cuerpo* Actual, Cuerpo *cuerp){
	if(n_Cuerpos==0){
		Root=new Cuerpo();
		Root->tipo=1;
		Root->indCuerpo=cuerp->indCuerpo;
		Root->masa=cuerp->masa;
		Root->PosX=cuerp->PosX;
		Root->PosY=cuerp->PosY;
		Root->PosZ=cuerp->PosZ;
		Root->VelX=cuerp->VelX;
		Root->VelY=cuerp->VelY;
		Root->VelZ=cuerp->VelZ;
		n_Cuerpos++;
	}
	else{
		if(Actual==NULL){
			Actual=new Cuerpo();
			Actual->tipo=1;
			Actual->indCuerpo=cuerp->indCuerpo;
			Actual->masa=cuerp->masa;
			Actual->PosX=cuerp->PosX;
			Actual->PosY=cuerp->PosY;
			Actual->PosZ=cuerp->PosZ;
			Actual->VelX=cuerp->VelX;
			Actual->VelY=cuerp->VelY;
			Actual->VelZ=cuerp->VelZ;
			n_Cuerpos++;
		}else {
			ObtenerHijo(Actual,cuerp);		
		}
	}
	return Actual;
}
/*

Cuerpo* GalaxiaQT::addCuerpo(Cuerpo* Actual, Cuerpo *cuerp){
	if(n_Cuerpos==0){
		Root=cuerp;
		n_Cuerpos++;
	}else{
		if (Actual==NULL){
			Actual=cuerp;
			n_Cuerpos++;
		}else{
			ObtenerHijo(Actual,cuerp);
		}
	}
}*/

int GalaxiaQT::CargarDub(char *fileD){
	int i=0;
	int contador=0;
	char linea[100];
	Cuerpo *TempCuerpos;
	FILE *file = fopen(fileD, "rb");
	if (!file){		
		return 0; 
	}
	int indice=fseek(file,0,SEEK_END);                
	int tamFile=ftell(file);
	rewind(file);

	//leer el archivo linea por linea
	while(i<MaxNumCuerpos){

		fgets(linea, 100, file);   

		//TempCuerpos=new Cuerpo();
		//TempCuerpos=(Cuerpo*)malloc(sizeof(Cuerpo)*10);
		TempCuerpos=new Cuerpo();

		TempCuerpos->indCuerpo=i;

		TempCuerpos->masa=atof(strtok( linea, " \n\t" ));	
		TempCuerpos->PosX=round(atof(strtok( NULL, " \n\t" )),4);	
		TempCuerpos->PosY=round(atof(strtok( NULL, " \n\t" )),4);	
		TempCuerpos->PosZ=round(atof(strtok( NULL, " \n\t" )),4);	
		TempCuerpos->VelX=round(atof(strtok( NULL, " \n\t" )),4);	
		TempCuerpos->VelY=round(atof(strtok( NULL, " \n\t" )),4);	
		TempCuerpos->VelZ=round(atof(strtok( NULL, " \n\t" )),4);	

		this->addCuerpo(Root,TempCuerpos);
		i++;
	}

	fclose(file);
	return true;	
}

double GalaxiaQT::round(double r,int n_digit){
	int n=pow(10.0,n_digit);
	r=((double)((int)(r*n+0.5)))/n;
	return(r);
}

double Distancia(Cuerpo *a, Cuerpo *b){
	dx=b->PosX-a->PosX;
	dy=b->PosY-a->PosY;;
	dz=b->PosZ-a->PosZ;;
	dist2=(dx*dx) + (dy*dy) + (dz*dz);
	dist=sqrt(dist2);
	return dist;
}

/*		0	|	1	|	2	|	3	|	4	|	5	|	6	|	7
			NE1	   SE1     SO1     NO1	   NE2	   SE2	    SO2	   NO2
	*/
double MayorDist(Cuerpo *a){
	double dist=0;
	if(a->NE1!= NULL && Distancia(a,a->NE1)>dist)
		dist=Distancia(a,a->NE1);
	if(a->SE1!= NULL && Distancia(a,a->SE1)>dist)
		dist=Distancia(a,a->SE1);
	if(a->SO1!= NULL && Distancia(a,a->SO1)>dist)
		dist=Distancia(a,a->SO1);
	if(a->NO1!= NULL && Distancia(a,a->NO1)>dist)
		dist=Distancia(a,a->NO1);
	if(a->NE2!= NULL && Distancia(a,a->NE2)>dist)
		dist=Distancia(a,a->NE2);
	if(a->SE2!= NULL && Distancia(a,a->SE2)>dist)
		dist=Distancia(a,a->SE2);
	if(a->SO2!= NULL && Distancia(a,a->SO2)>dist)
		dist=Distancia(a,a->SO2);
	if(a->NO2!= NULL && Distancia(a,a->NO2)>dist)
		dist=Distancia(a,a->NO2);
	return dist;
}

void GalaxiaQT::ActualizaPos(Cuerpo *c){
	while (c!=NULL)	{
		Gdtm=Gdt/c->masa;
		c->VelX=c->VelX+c->FueX *Gdtm;
		c->PosX=c->PosX+c->VelX*dt/4;
		c->VelY=c->VelY+c->FueY *Gdtm;
		c->PosY=c->PosY+c->VelY*dt/4;
		c->VelZ=c->VelZ+c->FueZ *Gdtm;
		c->PosZ=c->PosZ+c->VelZ*dt/4;
		c->FueX=c->FueY=c->FueZ=0;
		ActualizaPos(c->NE1);
		ActualizaPos(c->SE1);
		ActualizaPos(c->SO1);
		ActualizaPos(c->NO1);
		ActualizaPos(c->NE2);
		ActualizaPos(c->SE2);
		ActualizaPos(c->SO2);
		ActualizaPos(c->NO2);
		break;
	}
}

void GalaxiaQT::CalculoFuerza(Cuerpo *a,Cuerpo *b){

	while (b!=NULL)	{

		s=MayorDist(b);
		d=Distancia(a,b);

		if(s/d<0.45 || a==b){
			//cout<<"entro al ciclo"<<endl;
			CalculoFuerza(a,b->NE1);
			CalculoFuerza(a,b->SE1);
			CalculoFuerza(a,b->SO1);
			CalculoFuerza(a,b->NO1);
			CalculoFuerza(a,b->NE2);
			CalculoFuerza(a,b->SE2);
			CalculoFuerza(a,b->SO2);
			CalculoFuerza(a,b->NO2);
			break;
		}else{
			/*CALCULAMOS LA FUERZA*/
			//cout<<"calculo fuerza"<<endl;
			dx=b->PosX-a->PosX;
			dy=b->PosY-a->PosY;;
			dz=b->PosZ-a->PosZ;;
			dist2=(dx*dx) + (dy*dy) + (dz*dz);
			dist=sqrt(dist2);
			Fs=(b->masa*a->masa)/(dist*dist2);
			Fsdx=Fs*dx;
			Fsdy=Fs*dy;
			Fsdz=Fs*dz;
			b->FueX=b->FueX-Fsdx;
			a->FueX=a->FueX+Fsdx;
			b->FueY=b->FueY-Fsdy;
			a->FueY=a->FueY-Fsdy;
			b->FueZ=b->FueZ-Fsdz;
			a->FueZ=a->FueZ-Fsdz;
			break;
		}
	}
}



void GalaxiaQT::N_Body(Cuerpo *c){


	double G=0.00000000667;//cte gravitacional
	dt=5;//tiempo por iteracion
	double T=10;//tiempo total
	double k=T/dt;
	double temp=0;
	Gdt=G*dt;
	//cout<<"algoritmo NBODY "<<contador<<endl;
	contador++;

	Fx=Fy=Fz=0;
	while (c!=NULL){
		CalculoFuerza(c,c);
		N_Body(c->NE1);
		N_Body(c->SE1);
		N_Body(c->SO1);
		N_Body(c->NO1);
		N_Body(c->NE2);
		N_Body(c->SE2);
		N_Body(c->SO2);
		N_Body(c->NO2);
		break;
	}
}


GalaxiaQT::~GalaxiaQT(){		
}