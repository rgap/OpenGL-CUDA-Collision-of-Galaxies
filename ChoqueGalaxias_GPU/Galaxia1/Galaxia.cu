#include "hip/hip_runtime.h"
#include "Galaxia.cuh"

#include <iostream>

#include <hip/hip_runtime.h>
#include <>

#include "cuPrintf.cu"

#if __CUDA_ARCH__ < 200 	//Compute capability 1.x architectures
#define CUPRINTF cuPrintf 
#else						//Compute capability 2.x architectures
#define CUPRINTF(fmt, ...) printf("[%d, %d]:\t" fmt, \
								blockIdx.y*gridDim.x+blockIdx.x,\
								threadIdx.z*blockDim.x*blockDim.y+threadIdx.y*blockDim.x+threadIdx.x,\
								__VA_ARGS__)
#endif

using namespace std;

Galaxia::Galaxia(){
	//this->MaxNumCuerpos= 81920;  65536

	this->MaxNumCuerpos=81920;
	this->n_Cuerpos=0;

	cuerpos.FueX = new float [MaxNumCuerpos];
	cuerpos.FueY = new float [MaxNumCuerpos];
	cuerpos.FueZ = new float [MaxNumCuerpos];

	cuerpos.PosX = new float [MaxNumCuerpos];
	cuerpos.PosY = new float [MaxNumCuerpos];
	cuerpos.PosZ = new float [MaxNumCuerpos];

	cuerpos.VelX = new float [MaxNumCuerpos];
	cuerpos.VelY = new float [MaxNumCuerpos];
	cuerpos.VelZ = new float [MaxNumCuerpos];

	cuerpos.masa = new float [MaxNumCuerpos];
}


int Galaxia::CargarDub(char *fileD){

	int i=0;
	int contador=0;
	char linea[100];

	FILE *file = fopen(fileD, "rb");

	if (!file) return false; 

	int indice=fseek(file,0,SEEK_END);                
	int tamFile=ftell(file);

	rewind(file);

	//leer el archivo linea por linea
	while(i<MaxNumCuerpos){

		fgets(linea, 100, file);   

		cuerpos.masa[n_Cuerpos] = atof(strtok( linea, " \n\t" ));	
		cuerpos.PosX[n_Cuerpos] = round(atof(strtok( NULL, " \n\t" )),4);	
		cuerpos.PosY[n_Cuerpos] = round(atof(strtok( NULL, " \n\t" )),4);	
		cuerpos.PosZ[n_Cuerpos] = round(atof(strtok( NULL, " \n\t" )),4);	
		cuerpos.VelX[n_Cuerpos] = round(atof(strtok( NULL, " \n\t" )),4);	
		cuerpos.VelY[n_Cuerpos] = round(atof(strtok( NULL, " \n\t" )),4);	
		cuerpos.VelZ[n_Cuerpos] = round(atof(strtok( NULL, " \n\t" )),4);	

		n_Cuerpos++;

		i++;
	}

	fclose(file);

	return true;	
}

float Galaxia::round(float r,int n_digit){

	int n = pow(10.0,n_digit);
	r=((float)((int)(r*n+0.5)))/n;

	return r;
}

__global__ 
void N_Body_kernel_P1(int i,int MaxNumCuerpos,float *masa_GPU,float *PosX_GPU,float *PosY_GPU,float *PosZ_GPU,
					float *VelX_GPU,float *VelY_GPU,float *VelZ_GPU,float *FueX_GPU,float *FueY_GPU,float *FueZ_GPU,
					float Pxi,float Pyi,float Pzi,float Mi){

	float Fx,Fy,Fz;
	float dx,dy,dz,dist2,dist,Fs,Fsdx,Fsdy,Fsdz;

	int tempX;
	int tempY;
	int tempZ;

	Fx=Fy=Fz=0;

	int j = blockIdx.x*blockDim.x+threadIdx.x;
	
	//CUPRINTF("j = %d  \n",j);
	
	if(j>=(i+1) && j<MaxNumCuerpos){

		dx=PosX_GPU[j] - Pxi;
		dy=PosY_GPU[j] - Pyi;
		dz=PosZ_GPU[j] - Pzi;

		dist2=(dx*dx) + (dy*dy) + (dz*dz);

		dist=sqrtf(dist2);

		Fs=(masa_GPU[j] * Mi)/(dist*dist2);

		Fsdx=Fs*dx;
		Fsdy=Fs*dy;
		Fsdz=Fs*dz;

		//atomicDec(&FueX_GPU[j], Fsdx);

		Fx=Fx+Fsdx;

		//atomicDec(&FueY_GPU[j], Fsdy);

		Fy=Fy+Fsdy;

		//atomicDec(&FueZ_GPU[j], Fsdz);

		Fz=Fz+Fsdz;
	}

	__syncthreads();

}

/*
__global__ 
void N_Body_kernel_P2(int MaxNumCuerpos, Cuerpo *l_CuerposGPU,float G,float dt,float T){

	float Gdtm;
	float Gdt=G*dt;
	float temp=0;

	int i = blockIdx.x*blockDim.x+threadIdx.x;

	if(i<MaxNumCuerpos){

		Gdtm=Gdt/l_CuerposGPU[i].masa;

		l_CuerposGPU[i].VelX=temp=l_CuerposGPU[i].VelX+l_CuerposGPU[i].FueX *Gdtm;
		l_CuerposGPU[i].PosX=l_CuerposGPU[i].PosX+temp*dt;
		l_CuerposGPU[i].VelY=temp=l_CuerposGPU[i].VelY+l_CuerposGPU[i].FueY *Gdtm;
		l_CuerposGPU[i].PosY=l_CuerposGPU[i].PosY+temp*dt;
		l_CuerposGPU[i].VelZ=temp=l_CuerposGPU[i].VelZ+l_CuerposGPU[i].FueZ *Gdtm;
		l_CuerposGPU[i].PosZ=l_CuerposGPU[i].PosZ+temp*dt;
		l_CuerposGPU[i].FueX=l_CuerposGPU[i].FueY=l_CuerposGPU[i].FueZ=0;

	}
}
*/


void Galaxia::calcN_Body(int i,int MaxNumCuerpos,float Pxi,float Pyi,float Pzi,float Mi){
	
	cout<<endl<<endl;
	
	for(int i=0; i<10; ++i){
		cout<<cuerpos.masa[i]<<" ";
		cout<<cuerpos.PosX[i]<<" ";
		cout<<cuerpos.VelX[i]<<" ";
		cout<<cuerpos.FueX[i]<<" ";
		cout<<endl;
	}
	cout<<endl<<endl;

	/////////////////////////////////////////////////

	int numThreads = 256;


	int Grid_Dim_x=MaxNumCuerpos;	//Grid structure values

	int Block_Dim_x=numThreads;	//Block structure values


	dim3 Grid = dim3( Grid_Dim_x*1.0/numThreads );		// ceil(Grid_Dim_y*1.0/numThreads) 

	dim3 Block = dim3(Block_Dim_x);	// Block_Dim_y


	//////////// Memoria en device //////////////////

	size_t size = MaxNumCuerpos*sizeof(float);

	hipMalloc((void**)&masa_GPU, size);
	hipMalloc((void**)&PosX_GPU, size);
	hipMalloc((void**)&PosY_GPU, size);
	hipMalloc((void**)&PosZ_GPU, size);
	hipMalloc((void**)&VelX_GPU, size);
	hipMalloc((void**)&VelY_GPU, size);
	hipMalloc((void**)&VelZ_GPU, size);
	hipMalloc((void**)&FueX_GPU, size);
	hipMalloc((void**)&FueY_GPU, size);
	hipMalloc((void**)&FueZ_GPU, size);

	//COPIAR A DEVICE

	hipMemcpy(masa_GPU,cuerpos.masa,size,hipMemcpyHostToDevice);
	hipMemcpy(PosX_GPU,cuerpos.PosX,size,hipMemcpyHostToDevice);
	hipMemcpy(PosY_GPU,cuerpos.PosY,size,hipMemcpyHostToDevice);
	hipMemcpy(PosZ_GPU,cuerpos.PosZ,size,hipMemcpyHostToDevice);
	hipMemcpy(VelX_GPU,cuerpos.VelX,size,hipMemcpyHostToDevice);
	hipMemcpy(VelY_GPU,cuerpos.VelY,size,hipMemcpyHostToDevice);
	hipMemcpy(VelZ_GPU,cuerpos.VelZ,size,hipMemcpyHostToDevice);
	hipMemcpy(FueX_GPU,cuerpos.FueX,size,hipMemcpyHostToDevice);
	hipMemcpy(FueY_GPU,cuerpos.FueY,size,hipMemcpyHostToDevice);
	hipMemcpy(FueZ_GPU,cuerpos.FueZ,size,hipMemcpyHostToDevice);


	N_Body_kernel_P1<<<Grid,Block>>>(i,MaxNumCuerpos,masa_GPU,PosX_GPU,PosY_GPU,PosZ_GPU,VelX_GPU,VelY_GPU,VelZ_GPU,FueX_GPU,FueY_GPU,FueZ_GPU,Pxi,Pyi,Pzi,Mi);

	hipMemcpy(cuerpos.masa,masa_GPU,size,hipMemcpyDeviceToHost);
	hipMemcpy(cuerpos.PosX,PosX_GPU,size,hipMemcpyDeviceToHost);
	hipMemcpy(cuerpos.PosY,PosY_GPU,size,hipMemcpyDeviceToHost);
	hipMemcpy(cuerpos.PosZ,PosZ_GPU,size,hipMemcpyDeviceToHost);
	hipMemcpy(cuerpos.VelX,VelX_GPU,size,hipMemcpyDeviceToHost);
	hipMemcpy(cuerpos.VelY,VelY_GPU,size,hipMemcpyDeviceToHost);
	hipMemcpy(cuerpos.VelZ,VelZ_GPU,size,hipMemcpyDeviceToHost);
	hipMemcpy(cuerpos.FueX,FueX_GPU,size,hipMemcpyDeviceToHost);
	hipMemcpy(cuerpos.FueY,FueY_GPU,size,hipMemcpyDeviceToHost);
	hipMemcpy(cuerpos.FueZ,FueZ_GPU,size,hipMemcpyDeviceToHost);

	cout<<endl<<endl;
	for(int i=0; i<10; ++i){
		cout<<cuerpos.masa[i]<<" ";
		cout<<cuerpos.PosX[i]<<" ";
		cout<<cuerpos.VelX[i]<<" ";
		cout<<cuerpos.FueX[i]<<" ";
		cout<<endl;
	}
	cout<<endl<<endl;
	
	hipDeviceSynchronize();
}


/*
void Galaxia::N_Body(){

	float G=0.00000000667;//cte gravitacional
	float dt=5; //tiempo por iteracion
	float T=10; //tiempo total
	
	float temp=0;
	float Pxi,Pyi,Pzi,Fx,Fy,Fz,Mi;
	float dx,dy,dz,dist2,dist,Fs,Fsdx,Fsdy,Fsdz,Gdtm;
	float Gdt=G*dt;

	float k=T/dt;

	//cout<<"Algoritmo N body"<<endl;

	int t=0,i;

	while(t<k){

		for(int i=0;i<MaxNumCuerpos;i++){

			Pxi=cuerpos.PosX[i];
			Pyi=cuerpos.PosY[i];
			Pzi=cuerpos.PosZ[i];
			
			Mi=cuerpos.masa[i];
		
			calcN_Body(i,MaxNumCuerpos,Pxi,Pyi,Pzi,Mi);

			cuerpos.FueX[i]+=Fx;
			cuerpos.FueY[i]+=Fy;
			cuerpos.FueZ[i]+=Fz;			
		}

		for(i=0; i<MaxNumCuerpos; ++i){

			Gdtm = Gdt/cuerpos.masa[i];

			cuerpos.VelX[i] = temp = cuerpos.VelX[i] + cuerpos.FueX[i] *Gdtm;
			cuerpos.PosX[i] = cuerpos.PosX[i] + temp*dt;

			cuerpos.VelY[i] = temp = cuerpos.VelY[i] + cuerpos.FueY[i] *Gdtm;
			cuerpos.PosY[i] = cuerpos.PosY[i] + temp*dt;

			cuerpos.VelZ[i] = temp = cuerpos.VelZ[i] + cuerpos.FueZ[i] *Gdtm;
			cuerpos.PosZ[i] = cuerpos.PosZ[i] + temp*dt;

			cuerpos.FueX[i] = cuerpos.FueY[i] = cuerpos.FueZ[i] = 0;
		}

		t++;
	}

}
*/


void Galaxia::N_Body(){

	float G=0.00000000667;//cte gravitacional
	float dt=5; //tiempo por iteracion
	float T=10; //tiempo total
	
	float temp=0;
	float Pxi,Pyi,Pzi,Fx,Fy,Fz,Mi;
	float dx,dy,dz,dist2,dist,Fs,Fsdx,Fsdy,Fsdz,Gdtm;
	float Gdt=G*dt;

	float k=T/dt;

	//cout<<"Algoritmo N body"<<endl;

	int t=0,i;

	MaxNumCuerpos = 8000;

	while(t<k){

		for(int i=0;i<MaxNumCuerpos;i++){

			Pxi=cuerpos.PosX[i];
			Pyi=cuerpos.PosY[i];
			Pzi=cuerpos.PosZ[i];
			
			Mi=cuerpos.masa[i];
		
			//calcN_Body(i,MaxNumCuerpos,Pxi,Pyi,Pzi,Mi);
			
			for (int j=i+1;j<MaxNumCuerpos;j++){
				dx=cuerpos.PosX[j]-Pxi;
				dy=cuerpos.PosY[j]-Pyi;
				dz=cuerpos.PosZ[j]-Pzi;

				dist2=(dx*dx) + (dy*dy) + (dz*dz);
				dist=sqrt(dist2);

				Fs=(cuerpos.masa[j]*Mi)/(dist*dist2);

				Fsdx=Fs*dx;
				Fsdy=Fs*dy;
				Fsdz=Fs*dz;

				cuerpos.FueX[j]=cuerpos.FueX[j]-Fsdx;
				Fx=Fx+Fsdx;
				cuerpos.FueY[j]=cuerpos.FueY[j]-Fsdy;
				Fy=Fy+Fsdy;
				cuerpos.FueZ[j]=cuerpos.FueZ[j]-Fsdz;
				Fz=Fz+Fsdz;
			}

			cuerpos.FueX[i]+=Fx;
			cuerpos.FueY[i]+=Fy;
			cuerpos.FueZ[i]+=Fz;			
		}

		for(i=0; i<MaxNumCuerpos; ++i){

			Gdtm = Gdt/cuerpos.masa[i];

			cuerpos.VelX[i] = temp = cuerpos.VelX[i] + cuerpos.FueX[i] *Gdtm;
			cuerpos.PosX[i] = cuerpos.PosX[i] + temp*dt;

			cuerpos.VelY[i] = temp = cuerpos.VelY[i] + cuerpos.FueY[i] *Gdtm;
			cuerpos.PosY[i] = cuerpos.PosY[i] + temp*dt;

			cuerpos.VelZ[i] = temp = cuerpos.VelZ[i] + cuerpos.FueZ[i] *Gdtm;
			cuerpos.PosZ[i] = cuerpos.PosZ[i] + temp*dt;

			cuerpos.FueX[i] = cuerpos.FueY[i] = cuerpos.FueZ[i] = 0;
		}

		t++;
	}

}



Galaxia::~Galaxia(){
}